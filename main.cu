#include <iostream>
#include <iomanip>

#include "parse.h"
#include "sequential.h"
#include "util.cuh"
#include "kernels.cuh"

int main(int argc, char *argv[])
{
	program_options op = parse_arguments(argc,argv);
	int max_threads_per_block, number_of_SMs;
	choose_device(max_threads_per_block,number_of_SMs,op);
	
	graph g = parse(op.infile);

	std::cout << "Number of nodes: " << g.n << std::endl;
	std::cout << "Number of edges: " << g.m << std::endl;

	std::vector< std::vector<int> > d_cpu;	
	std::vector< std::vector<int> > d_gpu;
	std::vector< std::vector<unsigned long long> > sigma_cpu;
	std::vector< std::vector<unsigned long long> > sigma_gpu;

	hipEvent_t start,end;
	float CPU_time;
	std::vector<float> bc;
	if(op.verify) //Only run CPU code if verifying
	{
		start_clock(start,end);
		bc = bc_cpu(g,d_cpu,sigma_cpu);
		CPU_time = end_clock(start,end);
	}

	float GPU_time;
	std::vector<float> bc_g;
	start_clock(start,end);
	bc_g = bc_gpu(g,d_gpu,sigma_gpu,max_threads_per_block,number_of_SMs,op);
	GPU_time = end_clock(start,end);

	if(op.verify)
	{
		verify(g,bc,bc_g);
	}
	if(op.printBCscores)
	{
		g.print_BC_scores(bc_g,op.scorefile);
	}

	std::cout << std::setprecision(9);
	if(op.verify)
	{
		std::cout << "Time for CPU Algorithm: " << CPU_time << " s" << std::endl;
	}
	std::cout << "Time for GPU Algorithm: " << GPU_time << " s" << std::endl;
	
	delete[] g.R;
	delete[] g.C;
	delete[] g.F;

	return 0;
}
